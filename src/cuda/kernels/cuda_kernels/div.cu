
#include <hip/hip_runtime.h>
extern "C" __global__ void snn_div_kernel(
    float* a,
    float b,
    size_t numel
) {
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < numel) {
        a[i] = a[i] / b;
    }
}