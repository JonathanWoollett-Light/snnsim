
#include <hip/hip_runtime.h>
extern "C" __global__ void snn_abs_diff_kernel(
    float* a,
    float* b,
    float* c,
    size_t numel
) {
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < numel) {
        a[i] = a[i] + fabsf(b[i] - c[i]);
    }
}